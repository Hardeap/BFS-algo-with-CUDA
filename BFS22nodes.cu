#include "hip/hip_runtime.h"
/*
 * BFS.cu
 *
 *  Created on: Oct 15, 2017
 *      Author: singh-h18
 */


#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include<hip/hip_runtime_api.h>
#include<helper_timer.h>

#define number_of_nodes 30
#define MAX_THREADS_PER_BLOCK 512

typedef struct
{
	int f_index;
	int tnodes;
} Node;

__global__ void BFS_KERNEL(Node *d_vertices, int *d_edges, bool *d_frontier, bool *d_visited, int *d_cost,bool *update)
{

	int id = threadIdx.x + blockIdx.x * MAX_THREADS_PER_BLOCK;
	if (id > number_of_nodes)
		*update = false;


	if (d_frontier[id] == true && d_visited[id] == false)
	{
		printf("%d ", id);
		d_frontier[id] = false;
		d_visited[id] = true;
		__syncthreads();

		int f_index = d_vertices[id].f_index;
		int end = f_index + d_vertices[id].tnodes;
		for (int i = f_index; i < end; i++)
		{
			int nid = d_edges[i];

			if (d_visited[nid] == false)
			{
				d_cost[nid] = d_cost[id] + 1;
				d_frontier[nid] = true;
				*update = false;
			}

		}

	}

}




int main()
{




	 Node vertices[number_of_nodes];



	int edges[number_of_nodes];

	vertices[0].f_index = 0;
	vertices[0].tnodes = 2;

	vertices[1].f_index = 2;
	vertices[1].tnodes = 2;

	vertices[2].f_index = 4;
	vertices[2].tnodes = 2;

	vertices[3].f_index = 6;
	vertices[3].tnodes = 2;

	vertices[4].f_index = 8;
	vertices[4].tnodes = 2;


	vertices[5].f_index = 10;
	vertices[5].tnodes = 2;

	vertices[6].f_index = 12;
	vertices[6].tnodes = 2;

	vertices[7].f_index = 14;
	vertices[7].tnodes = 1;

	vertices[8].f_index = 15;
	vertices[8].tnodes = 1;

	vertices[9].f_index = 16;
	vertices[9].tnodes = 1;

	vertices[10].f_index = 17;
		vertices[10].tnodes = 1;


		vertices[11].f_index = 18;
		vertices[11].tnodes = 1;

		vertices[12].f_index = 19;
		vertices[12].tnodes = 1;

		vertices[13].f_index = 20;
		vertices[13].tnodes = 1;

		vertices[14].f_index = 21;
		vertices[14].tnodes = 1;

		vertices[15].f_index = 22;
		vertices[15].tnodes = 1;

		vertices[16].f_index = 23;
			vertices[16].tnodes = 1;


			vertices[17].f_index = 24;
			vertices[17].tnodes = 1;

			vertices[18].f_index = 25;
			vertices[18].tnodes = 1;

			vertices[19].f_index = 26;
			vertices[19].tnodes = 1;

			vertices[20].f_index = 27;
			vertices[20].tnodes = 1;

			vertices[21].f_index = 28;
			vertices[21].tnodes = 0;
	edges[0] = 1;
	edges[1] = 2;
	edges[2] = 3;
	edges[3] = 4;
	edges[4] = 5;
	edges[5] = 6;
		edges[6] = 7;
		edges[7] = 8;
		edges[8] = 9;
		edges[9] = 10;
		edges[10] = 11;
				edges[11] = 12;

				edges[12] = 13;
					edges[13] = 14;
					edges[14] = 15;
					edges[15] = 15;
					edges[16] = 16;
					edges[17] = 16;
					edges[18] = 17;
					edges[19] = 17;
					edges[20] = 18;
					edges[21] = 18;
					edges[22] = 19;
					edges[23] = 19;
					edges[24] = 20;
					edges[25] = 20;
					edges[26] = 21;
					edges[27] = 21;
			     	edges[28] = 0;


	bool h_frontier[number_of_nodes] = { false };
	bool h_visited[number_of_nodes] = { false };
	int h_cost[number_of_nodes] = { 0 };

	int source = 0;
	h_frontier[source] = true;
	int num_blocks = 1;
		int num_of_threads_per_block = number_of_nodes;


		if(num_blocks>MAX_THREADS_PER_BLOCK)
		{
			num_blocks = (int)ceil(number_of_nodes/(double)MAX_THREADS_PER_BLOCK);
			num_of_threads_per_block = MAX_THREADS_PER_BLOCK;
		}
	Node* d_vertices;
	hipMalloc((void**)&d_vertices, sizeof(Node)*number_of_nodes);
	hipMemcpy(d_vertices, vertices, sizeof(Node)*number_of_nodes, hipMemcpyHostToDevice);

	int* d_edges;
	hipMalloc((void**)&d_edges, sizeof(Node)*number_of_nodes);
	hipMemcpy(d_edges, edges, sizeof(Node)*number_of_nodes, hipMemcpyHostToDevice);

	bool* d_frontier;
	hipMalloc((void**)&d_frontier, sizeof(bool)*number_of_nodes);
	hipMemcpy(d_frontier, h_frontier, sizeof(bool)*number_of_nodes, hipMemcpyHostToDevice);

	bool* d_visited;
	hipMalloc((void**)&d_visited, sizeof(bool)*number_of_nodes);
	hipMemcpy(d_visited, h_visited, sizeof(bool)*number_of_nodes, hipMemcpyHostToDevice);

	int* d_cost;
	hipMalloc((void**)&d_cost, sizeof(int)*number_of_nodes);
	hipMemcpy(d_cost, h_cost, sizeof(int)*number_of_nodes, hipMemcpyHostToDevice);




	dim3  grid( num_blocks, 1, 1);
	dim3  threads( num_of_threads_per_block, 1, 1);


	float timer1 = 0.0f;

StopWatchInterface *timer = NULL;
sdkCreateTimer(&timer);
sdkStartTimer(&timer);

	bool update;
	bool* d_update;
	hipMalloc((void**)&d_update, sizeof(bool));
	printf("\n\n");
	int count = 0;

	printf("Order: \n");
	do {
		count++;
		update = true;
		hipMemcpy(d_update, &update, sizeof(bool), hipMemcpyHostToDevice);
		sdkStartTimer(&timer);
		BFS_KERNEL<<<grid, threads >>>(d_vertices, d_edges, d_frontier, d_visited, d_cost,d_update);
		hipMemcpy(&update, d_update , sizeof(bool), hipMemcpyDeviceToHost);
		sdkStopTimer(&timer);

		timer1 = sdkGetTimerValue(&timer);
		sdkResetTimer(&timer);
	} while (!update);
	hipMemcpy(h_cost, d_cost, sizeof(int)*number_of_nodes, hipMemcpyDeviceToHost);

	printf("\nTotal number of times kernel call : %d \n", count);
	printf("Total Execution time: %f (ms)", timer1);
sdkDeleteTimer(&timer);

	printf("\nCost: ");
	for (int i = 0; i<number_of_nodes; i++)
		printf( "%d    ", h_cost[i]);
	printf("\n");
 hipFree (d_visited);
 hipFree (d_vertices);
 hipFree (d_edges);
 hipFree (d_frontier);
 hipFree (d_cost);
 hipFree (d_update);


}
