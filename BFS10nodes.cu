#include "hip/hip_runtime.h"
/*
 * BFS.cu
 *
 *  Created on: Oct 15, 2017
 *      Author: singh-h18
 */


#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include<hip/hip_runtime_api.h>
#include<helper_timer.h>

#define number_of_nodes 12
#define MAX_THREADS_PER_BLOCK 512

typedef struct
{
	int f_index;
	int tnodes;
} Node;

__global__ void BFS_KERNEL(Node *d_vertices, int *d_edges, bool *d_frontier, bool *d_visited, int *d_cost,bool *update)
{

	int id = threadIdx.x + blockIdx.x * MAX_THREADS_PER_BLOCK;
	if (id > number_of_nodes)
		*update = false;


	if (d_frontier[id] == true && d_visited[id] == false)
	{
		printf("%d ", id);
		d_frontier[id] = false;
		d_visited[id] = true;
		__syncthreads();

		int f_index = d_vertices[id].f_index;
		int end = f_index + d_vertices[id].tnodes;
		for (int i = f_index; i < end; i++)
		{
			int nid = d_edges[i];

			if (d_visited[nid] == false)
			{
				d_cost[nid] = d_cost[id] + 1;
				d_frontier[nid] = true;
				*update = false;
			}

		}

	}

}




int main()
{




	 Node vertices[number_of_nodes];



	int edges[number_of_nodes];

	vertices[0].f_index = 0;
	vertices[0].tnodes = 2;

	vertices[1].f_index = 2;
	vertices[1].tnodes = 2;

	vertices[2].f_index = 4;
	vertices[2].tnodes = 2;

	vertices[3].f_index = 6;
	vertices[3].tnodes = 1;

	vertices[4].f_index = 7;
	vertices[4].tnodes = 1;


	vertices[5].f_index = 8;
	vertices[5].tnodes = 1;

	vertices[6].f_index = 9;
	vertices[6].tnodes = 1;

	vertices[7].f_index = 10;
	vertices[7].tnodes = 1;

	vertices[8].f_index = 11;
	vertices[8].tnodes = 1;

	vertices[9].f_index = 12;
	vertices[9].tnodes = 0;

	edges[0] = 1;
	edges[1] = 2;
	edges[2] = 3;
	edges[3] = 4;
	edges[4] = 5;
	edges[5] = 6;
		edges[6] = 7;
		edges[7] = 7;
		edges[8] = 8;
		edges[9] = 8;
		edges[10] = 9;
				edges[11] = 9;


	bool h_frontier[number_of_nodes] = { false };
	bool h_visited[number_of_nodes] = { false };
	int h_cost[number_of_nodes] = { 0 };

	int source = 0;
	h_frontier[source] = true;
	int num_blocks = 1;
		int num_of_threads_per_block = number_of_nodes;


		if(num_blocks>MAX_THREADS_PER_BLOCK)
		{
			num_blocks = (int)ceil(number_of_nodes/(double)MAX_THREADS_PER_BLOCK);
			num_of_threads_per_block = MAX_THREADS_PER_BLOCK;
		}
	Node* d_vertices;
	hipMalloc((void**)&d_vertices, sizeof(Node)*number_of_nodes);
	hipMemcpy(d_vertices, vertices, sizeof(Node)*number_of_nodes, hipMemcpyHostToDevice);

	int* d_edges;
	hipMalloc((void**)&d_edges, sizeof(Node)*number_of_nodes);
	hipMemcpy(d_edges, edges, sizeof(Node)*number_of_nodes, hipMemcpyHostToDevice);

	bool* d_frontier;
	hipMalloc((void**)&d_frontier, sizeof(bool)*number_of_nodes);
	hipMemcpy(d_frontier, h_frontier, sizeof(bool)*number_of_nodes, hipMemcpyHostToDevice);

	bool* d_visited;
	hipMalloc((void**)&d_visited, sizeof(bool)*number_of_nodes);
	hipMemcpy(d_visited, h_visited, sizeof(bool)*number_of_nodes, hipMemcpyHostToDevice);

	int* d_cost;
	hipMalloc((void**)&d_cost, sizeof(int)*number_of_nodes);
	hipMemcpy(d_cost, h_cost, sizeof(int)*number_of_nodes, hipMemcpyHostToDevice);




	dim3  grid( num_blocks, 1, 1);
	dim3  threads( num_of_threads_per_block, 1, 1);


	float timer1 = 0.0f;

StopWatchInterface *timer = NULL;
sdkCreateTimer(&timer);
sdkStartTimer(&timer);

	bool update;
	bool* d_update;
	hipMalloc((void**)&d_update, sizeof(bool));
	printf("\n\n");
	int count = 0;

	printf("Order: \n");
	do {
		count++;
		update = true;
		hipMemcpy(d_update, &update, sizeof(bool), hipMemcpyHostToDevice);
		sdkStartTimer(&timer);
		BFS_KERNEL<<<grid, threads >>>(d_vertices, d_edges, d_frontier, d_visited, d_cost,d_update);
		hipMemcpy(&update, d_update , sizeof(bool), hipMemcpyDeviceToHost);
		sdkStopTimer(&timer);

		timer1 = sdkGetTimerValue(&timer);
		sdkResetTimer(&timer);
	} while (!update);
	hipMemcpy(h_cost, d_cost, sizeof(int)*number_of_nodes, hipMemcpyDeviceToHost);

	printf("\nTotal number of times kernel call : %d \n", count);
	printf("Total Execution time: %f (ms)", timer1);
sdkDeleteTimer(&timer);

	printf("\nCost: ");
	for (int i = 0; i<number_of_nodes; i++)
		printf( "%d    ", h_cost[i]);
	printf("\n");
 hipFree (d_visited);
 hipFree (d_vertices);
 hipFree (d_edges);
 hipFree (d_frontier);
 hipFree (d_cost);
 hipFree (d_update);


}
