#include "hip/hip_runtime.h"
/*
 * BFS.cu
 *
 *  Created on: Oct 15, 2017
 *      Author: singh-h18
 */


#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include<hip/hip_runtime_api.h>
#include<helper_timer.h>

#define number_of_nodes 90
#define MAX_THREADS_PER_BLOCK 512

typedef struct
{
	int f_index;
	int tnodes;
} Node;

__global__ void BFS_KERNEL(Node *d_vertices, int *d_edges, bool *d_frontier, bool *d_visited, int *d_cost,bool *update)
{

	int id = threadIdx.x + blockIdx.x * MAX_THREADS_PER_BLOCK;
	if (id > number_of_nodes)
		*update = false;


	if (d_frontier[id] == true && d_visited[id] == false)
	{
		printf("%d ", id);
		d_frontier[id] = false;
		d_visited[id] = true;
		__syncthreads();

		int f_index = d_vertices[id].f_index;
		int end = f_index + d_vertices[id].tnodes;
		for (int i = f_index; i < end; i++)
		{
			int nid = d_edges[i];

			if (d_visited[nid] == false)
			{
				d_cost[nid] = d_cost[id] + 1;
				d_frontier[nid] = true;
				*update = false;
			}

		}

	}

}




int main()
{




	 Node vertices[number_of_nodes];



	int edges[number_of_nodes];

	vertices[0].f_index = 0;
	vertices[0].tnodes = 2;

	vertices[1].f_index = 2;
	vertices[1].tnodes = 2;

	vertices[2].f_index = 4;
	vertices[2].tnodes = 2;

	vertices[3].f_index = 6;
	vertices[3].tnodes = 3;

	vertices[4].f_index = 9;
	vertices[4].tnodes = 2;


	vertices[5].f_index = 11;
	vertices[5].tnodes = 2;

	vertices[6].f_index = 13;
	vertices[6].tnodes = 3;

	vertices[7].f_index = 16;
	vertices[7].tnodes = 3;

	vertices[8].f_index = 19;
	vertices[8].tnodes = 2;

	vertices[9].f_index = 21;
	vertices[9].tnodes = 2;

	vertices[10].f_index = 23;
		vertices[10].tnodes = 1;


		vertices[11].f_index = 24;
		vertices[11].tnodes = 1;

		vertices[12].f_index = 25;
		vertices[12].tnodes = 1;

		vertices[13].f_index = 26;
		vertices[13].tnodes = 1;

		vertices[14].f_index = 27;
		vertices[14].tnodes = 2;

		vertices[15].f_index = 29;
		vertices[15].tnodes = 2;

		vertices[16].f_index = 31;
			vertices[16].tnodes = 3;


			vertices[17].f_index = 34;
			vertices[17].tnodes = 1;

			vertices[18].f_index = 35;
			vertices[18].tnodes = 1;

			vertices[19].f_index = 36;
			vertices[19].tnodes = 1;

			vertices[20].f_index = 37;
			vertices[20].tnodes = 1;

			vertices[21].f_index = 38;
			vertices[21].tnodes = 1;
			vertices[22].f_index = 39;
					vertices[22].tnodes = 1;


					vertices[23].f_index = 40;
					vertices[23].tnodes = 1;

					vertices[24].f_index = 41;
					vertices[24].tnodes = 1;

					vertices[25].f_index = 42;
					vertices[25].tnodes = 1;

					vertices[26].f_index = 43;
					vertices[26].tnodes = 1;

					vertices[27].f_index = 44;
					vertices[27].tnodes = 1;

					vertices[28].f_index = 45;
						vertices[28].tnodes = 1;

						vertices[29].f_index = 46;
						vertices[29].tnodes = 1;

						vertices[30].f_index = 47;
						vertices[30].tnodes = 1;

						vertices[31].f_index = 48;
						vertices[31].tnodes = 1;

						vertices[32].f_index = 49;
						vertices[32].tnodes = 1;
						vertices[33].f_index = 50;
								vertices[33].tnodes = 1;


								vertices[34].f_index = 51;
								vertices[34].tnodes = 1;

								vertices[35].f_index = 52;
								vertices[35].tnodes = 1;

								vertices[36].f_index = 53;
								vertices[36].tnodes = 1;

								vertices[37].f_index = 54;
								vertices[37].tnodes = 1;

								vertices[38].f_index = 55;
								vertices[38].tnodes = 1;

								vertices[39].f_index = 56;
									vertices[39].tnodes = 1;


									vertices[40].f_index = 57;
									vertices[40].tnodes = 1;

									vertices[41].f_index = 58;
									vertices[41].tnodes = 1;

									vertices[42].f_index = 59;
									vertices[42].tnodes = 1;

									vertices[43].f_index = 60;
									vertices[43].tnodes = 1;

									vertices[44].f_index = 61;
									vertices[44].tnodes = 1;
									vertices[45].f_index = 62;
											vertices[45].tnodes = 1;


											vertices[46].f_index = 63;
											vertices[46].tnodes = 1;

											vertices[47].f_index = 64;
											vertices[47].tnodes = 1;

											vertices[48].f_index = 65;
											vertices[48].tnodes = 1;

											vertices[49].f_index = 66;
											vertices[49].tnodes = 1;

											vertices[50].f_index = 67;
											vertices[50].tnodes = 1;

											vertices[51].f_index = 68;
												vertices[51].tnodes = 1;


												vertices[52].f_index = 69;
												vertices[52].tnodes = 1;

												vertices[53].f_index = 70;
												vertices[53].tnodes = 1;

												vertices[54].f_index = 71;
												vertices[54].tnodes = 2;

												vertices[55].f_index = 73;
												vertices[55].tnodes = 2;

												vertices[56].f_index = 75;
												vertices[56].tnodes = 1;
												vertices[57].f_index = 76;
														vertices[57].tnodes = 1;


														vertices[58].f_index = 77;
														vertices[58].tnodes = 1;

														vertices[59].f_index = 78;
														vertices[59].tnodes = 1;

														vertices[60].f_index = 79;
														vertices[60].tnodes = 1;

														vertices[61].f_index = 80;
														vertices[61].tnodes = 1;

														vertices[62].f_index = 81;
														vertices[62].tnodes = 1;

														vertices[63].f_index = 82;
															vertices[63].tnodes = 0;



	edges[0] = 1;
	edges[1] = 2;
	edges[2] = 3;
	edges[3] = 4;
	edges[4] = 5;
	edges[5] = 6;
		edges[6] = 7;
		edges[7] = 8;
		edges[8] = 9;
		edges[9] = 10;
		edges[10] = 11;
				edges[11] = 12;

				edges[12] = 13;
					edges[13] = 14;
					edges[14] = 15;
					edges[15] = 16;
					edges[16] = 17;
					edges[17] = 18;
					edges[18] = 19;
					edges[19] = 20;
					edges[20] = 21;
					edges[21] = 22;
					edges[22] = 23;
					edges[23] = 24;
					edges[24] = 24;
					edges[25] = 25;
					edges[26] = 25;
					edges[27] = 26;
			     	edges[28] = 27;
			     	edges[29] = 28;
			     						edges[30] = 29;
			     						edges[31] = 30;
			     						edges[32] = 31;
			     						edges[33] = 32;
			     						edges[34] = 33;
			     						edges[35] = 34;
			     						edges[36] = 35;
			     						edges[37] = 36;
			     						edges[38] = 36;
			     						edges[39] = 37;
			     						edges[40] = 37;
			     						edges[41] = 38;
			     						edges[42] = 38;
			     						edges[43] = 39;
			     				     	edges[44] = 40;
			     				   	edges[45] = 41;
			     				   					edges[46] = 42;
			     				   					edges[47] = 43;
			     				   					edges[48] = 43;
			     				   					edges[49] = 44;
			     				   					edges[50] = 45;
			     				   					edges[51] = 45;
			     				   					edges[52] = 46;
			     				   					edges[53] = 46;
			     				   					edges[54] = 47;
			     				   					edges[55] = 47;
			     				   					edges[56] = 48;
			     				   					edges[57] = 48;
			     				   					edges[58] = 49;
			     				   					edges[59] = 49;
			     				   			     	edges[60] = 50;
			     				   			     	edges[61] = 50;

			     				   				edges[62] = 51;
			     				   								edges[63] = 51;
			     				   								edges[64] = 52;
			     				   								edges[65] = 52;
			     				   								edges[66] = 53;
			     				   								edges[67] = 53;
			     				   								edges[68] = 54;
			     				   								edges[69] = 55;
			     				   								edges[70] = 55;
			     				   								edges[71] = 56;
			     				   								edges[72] = 57;
			     				   								edges[73] = 58;
			     				   								edges[74] = 59;
			     				   								edges[75] = 60;
			     				   								edges[76] = 60;
			     				   						     	edges[77] = 61;
			     				   						     	edges[78] = 61;
			     				   						    edges[79] = 62;
			     				   						    	edges[80] = 62;
			      				   								edges[81] = 63;
			     				   						        edges[82] = 0;

	bool h_frontier[number_of_nodes] = { false };
	bool h_visited[number_of_nodes] = { false };
	int h_cost[number_of_nodes] = { 0 };

	int source = 0;
	h_frontier[source] = true;
	int num_blocks = 1;
		int num_of_threads_per_block = number_of_nodes;


		if(num_blocks>MAX_THREADS_PER_BLOCK)
		{
			num_blocks = (int)ceil(number_of_nodes/(double)MAX_THREADS_PER_BLOCK);
			num_of_threads_per_block = MAX_THREADS_PER_BLOCK;
		}
	Node* d_vertices;
	hipMalloc((void**)&d_vertices, sizeof(Node)*number_of_nodes);
	hipMemcpy(d_vertices, vertices, sizeof(Node)*number_of_nodes, hipMemcpyHostToDevice);

	int* d_edges;
	hipMalloc((void**)&d_edges, sizeof(Node)*number_of_nodes);
	hipMemcpy(d_edges, edges, sizeof(Node)*number_of_nodes, hipMemcpyHostToDevice);

	bool* d_frontier;
	hipMalloc((void**)&d_frontier, sizeof(bool)*number_of_nodes);
	hipMemcpy(d_frontier, h_frontier, sizeof(bool)*number_of_nodes, hipMemcpyHostToDevice);

	bool* d_visited;
	hipMalloc((void**)&d_visited, sizeof(bool)*number_of_nodes);
	hipMemcpy(d_visited, h_visited, sizeof(bool)*number_of_nodes, hipMemcpyHostToDevice);

	int* d_cost;
	hipMalloc((void**)&d_cost, sizeof(int)*number_of_nodes);
	hipMemcpy(d_cost, h_cost, sizeof(int)*number_of_nodes, hipMemcpyHostToDevice);




	dim3  grid( num_blocks, 1, 1);
	dim3  threads( num_of_threads_per_block, 1, 1);


	float timer1 = 0.0f;

StopWatchInterface *timer = NULL;
sdkCreateTimer(&timer);
sdkStartTimer(&timer);

	bool update;
	bool* d_update;
	hipMalloc((void**)&d_update, sizeof(bool));
	printf("\n\n");
	int count = 0;

	printf("Order: \n");
	do {
		count++;
		update = true;
		hipMemcpy(d_update, &update, sizeof(bool), hipMemcpyHostToDevice);
		sdkStartTimer(&timer);
		BFS_KERNEL<<<grid, threads >>>(d_vertices, d_edges, d_frontier, d_visited, d_cost,d_update);
		hipMemcpy(&update, d_update , sizeof(bool), hipMemcpyDeviceToHost);
		sdkStopTimer(&timer);

		timer1 = sdkGetTimerValue(&timer);
		sdkResetTimer(&timer);
	} while (!update);
	hipMemcpy(h_cost, d_cost, sizeof(int)*number_of_nodes, hipMemcpyDeviceToHost);

	printf("\nTotal number of times kernel call : %d \n", count);
	printf("Total Execution time: %f (ms)", timer1);
sdkDeleteTimer(&timer);

	printf("\nCost: ");
	for (int i = 0; i<number_of_nodes; i++)
		printf( "%d    ", h_cost[i]);
	printf("\n");
 hipFree (d_visited);
 hipFree (d_vertices);
 hipFree (d_edges);
 hipFree (d_frontier);
 hipFree (d_cost);
 hipFree (d_update);


}
